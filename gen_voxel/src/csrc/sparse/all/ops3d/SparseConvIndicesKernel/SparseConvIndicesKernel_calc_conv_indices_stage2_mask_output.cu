#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>

namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {

using TensorView = cumm::common::TensorView;
using TensorViewKernel = cumm::common::TensorViewKernel;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using ThrustLib = cumm::common::ThrustLib;
using ConvLocIter = csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;

__global__ void calc_conv_indices_stage2_mask_output(int* indice_pairs_bwd,
                                                     uint32_t* mask_bwd,
                                                     int num_indices_in, int kv)
{
    for (int input_index : tv::KernelLoopX<int>(num_indices_in)) {
        uint32_t mask = 0;
        for (int filter_offset = 0; filter_offset < kv; ++filter_offset) {
            auto val =
                indice_pairs_bwd[filter_offset * num_indices_in + input_index];
            mask |= (val != -1) << filter_offset;
        }
        mask_bwd[input_index] = mask;
    }
}

}  // namespace ops3d
}  // namespace all
}  // namespace sparse
}  // namespace csrc