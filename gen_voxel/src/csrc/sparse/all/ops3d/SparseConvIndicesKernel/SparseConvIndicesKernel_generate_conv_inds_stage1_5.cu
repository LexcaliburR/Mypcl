#include <csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = cumm::common::TensorView;
using TensorViewKernel = cumm::common::TensorViewKernel;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using ThrustLib = cumm::common::ThrustLib;
using ConvLocIter = csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;
int SparseConvIndicesKernel::generate_conv_inds_stage1_5(tv::Tensor indice_pairs_uniq, int64_t uniq_size, std::uintptr_t stream_int)   {
  
  thrust::device_ptr<int32_t> ptr_tr(indice_pairs_uniq.data_ptr<int32_t>());
  auto thrust_ctx = thrust::cuda::par.on(reinterpret_cast<hipStream_t>(stream_int));
  thrust::sort(thrust_ctx, ptr_tr, ptr_tr + uniq_size);
  auto new_end = thrust::unique(thrust_ctx, ptr_tr, ptr_tr + uniq_size);
  auto num_out_act = new_end - ptr_tr - 1;
  return num_out_act;
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc