#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = cumm::common::TensorView;
using TensorViewKernel = cumm::common::TensorViewKernel;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using ThrustLib = cumm::common::ThrustLib;
using ConvLocIter = csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;
int SparseConvIndicesKernel::generate_conv_inds_stage2(tv::Tensor indices, tv::Tensor hashdata, tv::Tensor indice_pairs, tv::Tensor indice_pairs_uniq, tv::Tensor out_inds, int num_out_act, int batch_size, tv::array<int, 3> output_dims, tv::array<int, 3> input_dims, tv::array<int, 3> ksize, tv::array<int, 3> stride, tv::array<int, 3> padding, tv::array<int, 3> dilation, bool transposed, std::uintptr_t stream_int)   {
  
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  // TODO stream
  // TODO handle num input == 0
  int kv = tv::arrayops::prod(ksize);
  TV_ASSERT_RT_ERR(kv == indice_pairs.dim(1), "error");
  // indice_pairs: [2, kv, indices.dim(0)]
  // indice_pairs_uniq: [indice_pairs.size() / 2 + 1]
  // out_inds: [MaxSize, 4]
  // auto timer = tv::CudaContextTimer<>();
  int64_t uniq_size = indice_pairs.size() / 2 + 1;
  TV_ASSERT_RT_ERR(indice_pairs_uniq.dim(0) >= num_out_act, "error");
  TV_ASSERT_RT_ERR(out_inds.dim(0) >= num_out_act && out_inds.dim(1) == 4, "error");
  tv::cuda::Launch launcher_num_act_in(indices.dim(0), custream);
  launcher_num_act_in.blocks.y = kv;
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  ConvLocIter loc_iter(problem);
  // TODO handle invalid num_out_act
  indice_pairs_uniq = indice_pairs_uniq.slice_first_axis(0, num_out_act);
  tv::cuda::Launch lanucher_build_hash(num_out_act, custream);
  using V = int32_t;
  using KeyType = int32_t;
  constexpr KeyType kEmptyKey = std::numeric_limits<KeyType>::max();
  using table_t =
      tv::hash::LinearHashTable<KeyType, V, tv::hash::Murmur3Hash<KeyType>,
                                  kEmptyKey, false>;
  using pair_t = typename table_t::value_type;
  TV_ASSERT_RT_ERR(hashdata.dim(0) >= num_out_act, "hash size not enough");
  table_t hash = table_t(hashdata.data_ptr<pair_t>(), hashdata.dim(0));
  hash.clear(custream);
  lanucher_build_hash(build_conv_hash_table<table_t>, hash, 
      out_inds.data_ptr<int>(), indice_pairs_uniq.data_ptr<const int32_t>(), 
      loc_iter.layout_npq, num_out_act);
  launcher_num_act_in(calc_conv_indices_stage2<table_t>, hash, 
      indice_pairs[1].data_ptr<int>(), indices.dim(0), 
      indice_pairs.dim(2));
  return num_out_act;
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc