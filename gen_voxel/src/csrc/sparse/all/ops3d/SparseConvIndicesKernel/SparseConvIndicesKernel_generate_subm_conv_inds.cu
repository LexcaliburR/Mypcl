#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>

namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {

using TensorView = cumm::common::TensorView;
using TensorViewKernel = cumm::common::TensorViewKernel;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using ThrustLib = cumm::common::ThrustLib;
using ConvLocIter = csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;

int SparseConvIndicesKernel::generate_subm_conv_inds(
    tv::Tensor indices, tv::Tensor hashdata, tv::Tensor indice_pairs,
    tv::Tensor out_inds, tv::Tensor indice_num_per_loc, int batch_size,
    tv::array<int, 3> input_dims, tv::array<int, 3> ksize,
    tv::array<int, 3> dilation, tv::Tensor indice_pair_mask, bool backward,
    std::uintptr_t stream_int)
{
    auto custream = reinterpret_cast<hipStream_t>(stream_int);
    auto ctx = tv::Context();
    ctx.set_cuda_stream(custream);
    if (!indice_pair_mask.empty()) {
        TV_ASSERT_INVALID_ARG(tv::arrayops::prod(ksize) < 32,
                              "for now only support 32bit mask");
    }
    // TODO stream
    // TODO handle num input == 0
    tv::array<int, 3> stride, padding;
    for (int i = 0; i < 3; ++i) {
        TV_ASSERT_RT_ERR(ksize[i] % 2 == 1, "subm only support odd ksize");
        stride[i] = 1;
        padding[i] = (ksize[i] / 2) * dilation[i];
    }
    int kv = tv::arrayops::prod(ksize);
    TV_ASSERT_RT_ERR(kv == indice_pairs.dim(1), "error");
    // indice_pairs: [2, kv, indices.dim(0)]
    // out_inds: [MaxSize, 4]
    // auto timer = tv::CudaContextTimer<>();
    TV_ASSERT_RT_ERR(indice_num_per_loc.dim(0) == kv, "error");
    tv::cuda::Launch launcher_num_act_in(indices.dim(0), custream);
    launcher_num_act_in.blocks.y = (kv / 2) + 1;
    // launcher_num_act_in.blocks.y = kv;
    TV_ASSERT_RT_ERR(
        tv::arrayops::prod(input_dims) <= std::numeric_limits<int32_t>::max(),
        "kernel volume must smaller than max value of int32_t");
    ConvProblem problem(batch_size,
                        1,
                        1,
                        input_dims,
                        input_dims,
                        ksize,
                        padding,
                        stride,
                        dilation);
    ConvLocIter loc_iter(problem);
    tv::cuda::Launch lanucher_build_hash(indices.dim(0), custream);
    using V = int32_t;
    using KeyType = int32_t;
    constexpr KeyType kEmptyKey = std::numeric_limits<KeyType>::max();
    using table_t = tv::hash::LinearHashTable<KeyType,
                                              V,
                                              tv::hash::Murmur3Hash<KeyType>,
                                              kEmptyKey,
                                              false>;
    using pair_t = typename table_t::value_type;
    TV_ASSERT_RT_ERR(hashdata.dim(0) >= indices.dim(0), "hash size not enough");
    table_t hash = table_t(hashdata.data_ptr<pair_t>(), hashdata.dim(0));
    hash.clear(custream);
    // tv::ssprint("clear hash time", hashdata.dim(0), timer.report() / 1000.0);
    lanucher_build_hash(build_subm_conv_hash_table<table_t>,
                        hash,
                        indices.data_ptr<const int>(),
                        loc_iter.layout_npq,
                        indices.dim(0));
    // tv::ssprint("build_hash time", timer.report() / 1000.0);
    if (!indice_pair_mask.empty()) {
        TV_ASSERT_INVALID_ARG(indice_pair_mask.ndim() == 2, "error");
        if (indice_pair_mask.dim(0) == 2) {
            auto mask_0 = indice_pair_mask[0];
            tv::cuda::Launch lanucher_fill(mask_0.size(), custream);
            lanucher_fill(cudakers::fill_kernel<uint32_t>,
                          mask_0.data_ptr<uint32_t>(),
                          (1 << (kv / 2)),
                          mask_0.size());
            indice_pair_mask[1].zero_(ctx);
            auto kernel = &calc_subm_conv_indices_split_mask<table_t>;
            launcher_num_act_in(kernel,
                                loc_iter,
                                hash,
                                indices.data_ptr<int>(),
                                indice_pairs.data_ptr<int>(),
                                indice_pair_mask[0].data_ptr<uint32_t>(),
                                indice_pair_mask[1].data_ptr<uint32_t>(),
                                indices.dim(0),
                                indice_pairs.dim(2),
                                kv);
        } else {
            tv::cuda::Launch lanucher_fill(indice_pair_mask.size(), custream);
            lanucher_fill(cudakers::fill_kernel<uint32_t>,
                          indice_pair_mask.data_ptr<uint32_t>(),
                          (1 << (kv / 2)),
                          indice_pair_mask.size());
            TV_ASSERT_RT_ERR(indice_pair_mask.dim(0) == 1, "error");
            launcher_num_act_in(calc_subm_conv_indices_mask<table_t>,
                                loc_iter,
                                hash,
                                indices.data_ptr<int>(),
                                indice_pairs.data_ptr<int>(),
                                indice_pair_mask.data_ptr<uint32_t>(),
                                indices.dim(0),
                                indice_pairs.dim(2),
                                kv);
        }
    } else {
        launcher_num_act_in(calc_subm_conv_indices<table_t>,
                            loc_iter,
                            hash,
                            indices.data_ptr<int>(),
                            indice_pairs.data_ptr<int>(),
                            indice_num_per_loc.data_ptr<int>(),
                            indices.dim(0),
                            indice_pairs.dim(2),
                            kv);
    }
    // tv::ssprint("gem subm conv inds time", timer.report() / 1000.0);
    return indices.dim(0);
}
}  // namespace ops3d
}  // namespace all
}  // namespace sparse
}  // namespace csrc