#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = cumm::common::TensorView;
using TensorViewKernel = cumm::common::TensorViewKernel;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using ThrustLib = cumm::common::ThrustLib;
using ConvLocIter = csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;
int SparseConvIndicesKernel::generate_conv_inds_stage2_mask(tv::Tensor indices, tv::Tensor hashdata, tv::Tensor indice_pairs_fwd, tv::Tensor indice_pairs_bwd, tv::Tensor indice_pairs_uniq, tv::Tensor out_inds, tv::Tensor mask_fwd, tv::Tensor mask_bwd, int num_out_act, int batch_size, tv::array<int, 3> output_dims, tv::array<int, 3> input_dims, tv::array<int, 3> ksize, tv::array<int, 3> stride, tv::array<int, 3> padding, tv::array<int, 3> dilation, bool transposed, std::uintptr_t stream_int)   {
  
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  // TODO stream
  // TODO handle num input == 0
  int kv = tv::arrayops::prod(ksize);
  // indice_pairs_bwd: [kv, indices.dim(0)]
  // indice_pairs_fwd: [kv, out_inds.dim(0)]
  auto ctx = tv::Context();
  ctx.set_cuda_stream(custream);
  // out_inds: [MaxSize, 4]
  // auto timer = tv::CudaContextTimer<>();
  tv::check_shape(indice_pairs_bwd, {kv, indices.dim(0)});
  tv::check_shape(indice_pairs_fwd, {kv, num_out_act});
  tv::check_shape(out_inds, {num_out_act, 4});
  tv::cuda::Launch launcher_num_act_in(indices.dim(0), custream);
  launcher_num_act_in.blocks.y = kv;
  tv::cuda::Launch launcher_num_act_in_no_y(indices.dim(0), custream);
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  ConvLocIter loc_iter(problem);
  // TODO handle invalid num_out_act
  indice_pairs_uniq = indice_pairs_uniq.slice_first_axis(0, num_out_act);
  tv::cuda::Launch lanucher_build_hash(num_out_act, custream);
  using V = int32_t;
  using KeyType = int32_t;
  constexpr KeyType kEmptyKey = std::numeric_limits<KeyType>::max();
  using table_t =
      tv::hash::LinearHashTable<KeyType, V, tv::hash::Murmur3Hash<KeyType>,
                                  kEmptyKey, false>;
  using pair_t = typename table_t::value_type;
  TV_ASSERT_RT_ERR(hashdata.dim(0) >= num_out_act, "hash size not enough");
  table_t hash = table_t(hashdata.data_ptr<pair_t>(), hashdata.dim(0));
  hash.clear(custream);
  lanucher_build_hash(build_conv_hash_table<table_t>, hash, 
      out_inds.data_ptr<int>(), indice_pairs_uniq.data_ptr<const int32_t>(), 
      loc_iter.layout_npq, num_out_act);
  if (!mask_bwd.empty()){
      // auto timer = tv::CudaContextTimer<>();
      launcher_num_act_in(calc_conv_indices_stage2_mask<table_t>, hash, 
          indice_pairs_fwd.data_ptr<int>(), indice_pairs_bwd.data_ptr<int>(), 
          mask_fwd.data_ptr<uint32_t>(), mask_bwd.data_ptr<uint32_t>(),
          indice_pairs_bwd.dim(1), indice_pairs_fwd.dim(1));
      // tv::ssprint("calc_conv_indices_stage2_mask", timer.report() / 1000.0);
      launcher_num_act_in_no_y(calc_conv_indices_stage2_mask_output, indice_pairs_bwd.data_ptr<int>(), 
          mask_bwd.data_ptr<uint32_t>(),
          indice_pairs_bwd.dim(1), kv);
      // tv::ssprint("calc_conv_indices_stage2_mask_output", timer.report() / 1000.0);
      if (mask_fwd.dim(0) == 2){
          mask_fwd[1].copy_(mask_fwd[0], ctx);
      }
      if (mask_bwd.dim(0) == 2){
          mask_bwd[1].copy_(mask_bwd[0], ctx);
      }
  }else{
      launcher_num_act_in(calc_conv_indices_stage2_inference_mask<table_t>, hash, 
          indice_pairs_fwd.data_ptr<int>(), indice_pairs_bwd.data_ptr<int>(), 
          mask_fwd.data_ptr<uint32_t>(),
          indice_pairs_bwd.dim(1), indice_pairs_fwd.dim(1));
      if (mask_fwd.dim(0) == 2){
          mask_fwd[1].copy_(mask_fwd[0], ctx);
      }
  }
  return num_out_act;
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc