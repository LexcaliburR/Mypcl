#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = cumm::common::TensorView;
using TensorViewKernel = cumm::common::TensorViewKernel;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using ThrustLib = cumm::common::ThrustLib;
using ConvLocIter = csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;
__global__ void clean_indices_uniq(int32_t* indice_pairs_for_uniq, int32_t size)   {
  
  for (int32_t i : tv::KernelLoopX<int32_t>(size)) {
      indice_pairs_for_uniq[i] = std::numeric_limits<int32_t>::max();
  }
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc