#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {

using TensorView = cumm::common::TensorView;
using TensorViewKernel = cumm::common::TensorViewKernel;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using ThrustLib = cumm::common::ThrustLib;
using ConvLocIter = csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;

__global__ void calc_conv_indices_stage1(
    ConvLocIter loc_iter, const int* indices_in, int32_t* indice_pairs,
    int32_t* indice_pairs_for_uniq, int* indice_num_per_loc, int num_indices_in,
    int indices_pair_size, int RS, bool transposed)
{
    int filter_offset = blockIdx.y;
    loc_iter.set_filter_offset(filter_offset);
    int indices_pair_size_mul_RS = indices_pair_size * RS;
    int filter_offset_mul_indices_pair_size = filter_offset * indices_pair_size;
    for (int i : tv::KernelLoopX<int>(num_indices_in)) {
        tv::array<int, 4> npq_offset;
        bool valid;
        if (transposed) {
            valid = loc_iter.query_nhw_out(indices_in + i * 4, npq_offset);
        } else {
            valid = loc_iter.query_npq(indices_in + i * 4, npq_offset);
        }
        if (valid) {
            int old_num =
                tv::cuda::atomicAggInc(indice_num_per_loc + filter_offset);
            int32_t offset = loc_iter.layout_npq(npq_offset);
            if (old_num < indices_pair_size) {
                indice_pairs[filter_offset_mul_indices_pair_size + old_num] = i;
                indice_pairs[indices_pair_size_mul_RS +
                             filter_offset_mul_indices_pair_size + old_num] =
                    offset;
                indice_pairs_for_uniq[filter_offset_mul_indices_pair_size +
                                      old_num] = offset;
            }
        }
    }
}

}  // namespace ops3d
}  // namespace all
}  // namespace sparse
}  // namespace csrc