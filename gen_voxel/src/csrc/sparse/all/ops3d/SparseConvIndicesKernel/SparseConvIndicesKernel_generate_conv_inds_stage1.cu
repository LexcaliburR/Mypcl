#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>

namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = cumm::common::TensorView;
using TensorViewKernel = cumm::common::TensorViewKernel;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using ThrustLib = cumm::common::ThrustLib;
using ConvLocIter = csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;
void SparseConvIndicesKernel::generate_conv_inds_stage1(tv::Tensor indices, tv::Tensor indice_pairs, tv::Tensor indice_pairs_uniq, tv::Tensor indice_num_per_loc, int batch_size, tv::array<int, 3> output_dims, tv::array<int, 3> input_dims, tv::array<int, 3> ksize, tv::array<int, 3> stride, tv::array<int, 3> padding, tv::array<int, 3> dilation, bool transposed, std::uintptr_t stream_int)   {
  
  // TODO stream
  // TODO handle num input == 0
  int kv = tv::arrayops::prod(ksize);
  TV_ASSERT_RT_ERR(kv == indice_pairs.dim(1), "error");
  TV_ASSERT_RT_ERR(tv::arrayops::prod(input_dims) <= std::numeric_limits<int32_t>::max(), 
      "kernel volume must smaller than max value of int32_t");
  // indice_pairs: [2, kv, indices.dim(0)]
  // indice_pairs_uniq: [indice_pairs.size() / 2 + 1]
  tv::check_shape(indice_pairs, {2, kv, indices.dim(0)});
  tv::check_shape(indice_num_per_loc, {kv});
  int64_t uniq_size = indice_pairs.size() / 2 + 1;
  TV_ASSERT_RT_ERR(indice_pairs_uniq.dim(0) >= uniq_size, "error");
  TV_ASSERT_RT_ERR(indice_num_per_loc.dim(0) == kv, "error");
  int64_t expected_out_size = indices.dim(0) * kv;
  tv::cuda::Launch launcher_num_act_in(indices.dim(0), reinterpret_cast<hipStream_t>(stream_int));
  // tv::cuda::Launch launcher_num_act_in_2(indices.dim(0));
  launcher_num_act_in.blocks.y = kv;
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  ConvLocIter loc_iter(problem);
  tv::cuda::Launch launcher_clean_uniq(uniq_size, reinterpret_cast<hipStream_t>(stream_int));
  launcher_clean_uniq(clean_indices_uniq, indice_pairs_uniq.data_ptr<int32_t>(), uniq_size);
  launcher_num_act_in(calc_conv_indices_stage1, loc_iter, indices.data_ptr<const int>(), 
      indice_pairs.data_ptr<int32_t>(), 
      indice_pairs_uniq.data_ptr<int32_t>(), indice_num_per_loc.data_ptr<int>(), indices.dim(0),
      indice_pairs.dim(2), kv, transposed);
  // thrust::device_ptr<int32_t> ptr_tr(indice_pairs_uniq.data_ptr<int32_t>());
  // auto thrust_ctx = thrust::cuda::par.on(reinterpret_cast<hipStream_t>(stream_int));
  // thrust::sort(thrust_ctx, ptr_tr, ptr_tr + uniq_size);
  // auto new_end = thrust::unique(thrust_ctx, ptr_tr, ptr_tr + uniq_size);
  // auto num_out_act = new_end - ptr_tr - 1;
  // return num_out_act;
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc