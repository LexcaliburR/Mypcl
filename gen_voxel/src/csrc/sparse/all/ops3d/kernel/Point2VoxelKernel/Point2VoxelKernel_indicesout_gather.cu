#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/kernel/Point2VoxelKernel.h>
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
namespace kernel {

using TensorView = cumm::common::TensorView;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using Layout = csrc::sparse::all::ops3d::layout_ns::TensorGeneric;

__global__ void indicesout_gather(int* indices, int* indices_out,
                                  int num_voxels, int num_coords,
                                  int num_coords_out)
{
    for (int i : tv::KernelLoopX<int>(num_voxels)) {
        for (int j = 0; j < num_coords; ++j) {
            indices_out[1 + i * num_coords_out + j] =
                indices[i * num_coords + j];
        }
    }
}

}  // namespace kernel
}  // namespace ops3d
}  // namespace all
}  // namespace sparse
}  // namespace csrc