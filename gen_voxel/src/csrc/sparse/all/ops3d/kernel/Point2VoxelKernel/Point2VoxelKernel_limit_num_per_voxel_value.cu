#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/kernel/Point2VoxelKernel.h>
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
namespace kernel {

using TensorView = cumm::common::TensorView;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using Layout = csrc::sparse::all::ops3d::layout_ns::TensorGeneric;

__global__ void limit_num_per_voxel_value(int* num_per_voxel, int num_voxels,
                                          int num_points_per_voxel)
{
    for (int i : tv::KernelLoopX<int>(num_voxels)) {
        int count = min(num_points_per_voxel, num_per_voxel[i]);
        num_per_voxel[i] = count;
    }
}

}  // namespace kernel
}  // namespace ops3d
}  // namespace all
}  // namespace sparse
}  // namespace csrc