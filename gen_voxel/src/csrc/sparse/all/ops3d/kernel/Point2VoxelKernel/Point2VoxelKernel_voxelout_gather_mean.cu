#include "hip/hip_runtime.h"
#include <csrc/sparse/all/ops3d/kernel/Point2VoxelKernel.h>
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
namespace kernel {

using TensorView = cumm::common::TensorView;
using TensorViewHashKernel = cumm::common::TensorViewHashKernel;
using Layout = csrc::sparse::all::ops3d::layout_ns::TensorGeneric;

__global__ void voxelout_gather_mean(float* voxels, float* voxels_out,
                                     int* num_per_voxel, int num_voxels,
                                     int num_points_per_voxel,
                                     int num_voxel_features)
{
    int voxel_stride = num_points_per_voxel * num_voxel_features;
    int voxel_out_stride = num_voxel_features;

    for (int i : tv::KernelLoopX<int>(num_voxels)) {
        int count = min(num_points_per_voxel, num_per_voxel[i]);
        num_per_voxel[i] = count;
        for (int j = 0; j < num_voxel_features; ++j) {
            auto voxel_ptr = voxels + i * voxel_stride + j;
            auto voxel_out_ptr = voxels_out + i * voxel_out_stride + j;
            float sum_val = 0;
            for (int k = 0; k < count; ++k) {
                sum_val += voxel_ptr[0];
                voxel_ptr += num_voxel_features;
            }
            sum_val = count == 0 ? 0 : sum_val / count;
            voxel_out_ptr[0] = sum_val;
        }
    }
}

}  // namespace kernel
}  // namespace ops3d
}  // namespace all
}  // namespace sparse
}  // namespace csrc